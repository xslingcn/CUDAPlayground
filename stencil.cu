#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>

// https://imgur.com/a/juCkz2a

__global__ void plain_conv_1d(float *N, float *M, float *P, int size, int kernelSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float pValue = 0;
    int iStart = i - kernelSize / 2;

    if (i < size)
    {
        for (int j = 0; j < kernelSize; j++)
        {
            if (iStart + j < 0 || iStart + j >= size)
                continue;
            pValue += N[iStart + j] * M[j];
        }
    }

    P[i] = pValue;
}

__global__ void plain_conv_2d(float *N, float *M, float *P, int width, int height, int kernelSize)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float pValue = 0;
    int colStart = col - kernelSize / 2;
    int rowStart = row - kernelSize / 2;

    if (col < width && row < height)
    {
        for (int j = 0; j < kernelSize; j++)
        {
            for (int k = 0; k < kernelSize; k++)
            {
                if (colStart + k < 0 || colStart + k >= width)
                    continue;
                if (rowStart + j < 0 || rowStart + j >= height)
                    continue;
                pValue += N[(rowStart + j) * width + colStart + k] * M[j * kernelSize + k];
            }
        }
    }

    P[row * width + col] = pValue;
}

#define O_TILE_SIZE 1020
__global__ void tiled_conv_1d(float *N, float *M, float *P, int size, int kernelSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    __shared__ float Ns[O_TILE_SIZE + 4];   // O_TILE_WIDTH + kernelSize - 1, assumeing 5

    int nStart = blockIdx.x * blockDim.x - kernelSize / 2; // index_o - n, n=kernelSize/2
    if (nStart + tx >= 0 && nStart + tx < size)
    {
        Ns[tx] = N[nStart + tx];
    }
    else
    {
        Ns[tx] = 0.0f;
    }
    __syncthreads();

    float pValue = 0;
    if (i < size)
    {
        for (int j = 0; j < kernelSize; j++)
        {
            pValue += Ns[tx + j] * M[j];
        }
        P[i] = pValue;
    }
}

#define O_TILE_SIZE_2D 28
__global__ void tiled_conv_2d(float *N, const float* __restrict__ M, float *P, int width, int height, int kernelSize)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int col_o = blockIdx.x * O_TILE_SIZE_2D + tx;
    int row_o = blockIdx.y * O_TILE_SIZE_2D + ty;
    int row_i = row_o - kernelSize / 2;
    int col_i = col_o - kernelSize / 2;

    __shared__ float Ns[O_TILE_SIZE_2D + 4][O_TILE_SIZE_2D + 4];

    if(row_i >= 0 && row_i < height && col_i >= 0 && col_i < width){
        Ns[ty][tx] = N[row_i * width + col_i];
    } else {
        Ns[ty][tx] = 0.0f;
    }
    __syncthreads();

    float pValue = 0;
    if (tx < O_TILE_SIZE_2D && ty < O_TILE_SIZE_2D)
    {
        for (int i = 0; i < kernelSize; i++)
        {
            for (int j = 0; j < kernelSize; j++)
            {
                pValue += Ns[ty + i][tx + j] * M[i * kernelSize + j];
            }
        }
    }
    if(row_o < height && col_o < width){
        P[row_o * width + col_o] = pValue;
    }
}

void test_simple_conv_1d()
{
    // int size = 7;
    int size = 5000;
    int kernelSize = 5;

    // float h_N[size] = {1, 2, 3, 4, 5, 6, 7};
    float h_N[size];
    for (int i = 0; i < size; i++)
    {
        h_N[i] = static_cast<float>(i);
    }
    float h_M[kernelSize] = {3, 4, 5, 4, 3};
    float h_P[size];

    float *d_N, *d_M, *d_P;

    hipMalloc((void **)&d_N, size * sizeof(float));
    hipMalloc((void **)&d_M, kernelSize * sizeof(float));
    hipMalloc((void **)&d_P, size * sizeof(float));

    hipMemcpy(d_N, h_N, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, kernelSize * sizeof(float), hipMemcpyHostToDevice);

    plain_conv_1d<<<ceil(size / 256.0), 256.0>>>(d_N, d_M, d_P, size, kernelSize);

    hipMemcpy(h_P, d_P, size * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < size; i++)
    // {
    //     printf("%f ", h_P[i]);
    // }
    // printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

void test_simple_conv_2d()
{
    int width = 5, height = 5;
    int kernelSize = 3;
    float h_N[width * height];
    float h_M[kernelSize * kernelSize] = {3, 4, 5, 6, 7, 6, 5, 4, 3};
    float h_P[width * height];

    for (int i = 0; i < width * height; i++)
    {
        h_N[i] = static_cast<float>(i);
    }

    float *d_N, *d_M, *d_P;

    hipMalloc((void **)&d_N, width * height * sizeof(float));
    hipMalloc((void **)&d_M, kernelSize * kernelSize * sizeof(float));
    hipMalloc((void **)&d_P, width * height * sizeof(float));

    hipMemcpy(d_N, h_N, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(ceil(width / 32.0), ceil(height / 32.0));
    plain_conv_2d<<<numBlocks, threadsPerBlock>>>(d_N, d_M, d_P, width, height, kernelSize);

    hipMemcpy(h_P, d_P, width * height * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < width * height; i++)
    // {
    //     printf("%f ", h_P[i]);
    // }
    // printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

void test_tiled_conv_1d (){
    int size = 5000;
    int kernelSize = 5;
    float h_N[size];
    float h_M[kernelSize] = {3, 4, 5, 4, 3};
    float h_P[size];

    for (int i = 0; i < size; i++)
    {
        h_N[i] = static_cast<float>(i);
    }

    float *d_N, *d_M, *d_P;

    hipMalloc((void **)&d_N, size * sizeof(float));
    hipMalloc((void **)&d_M, kernelSize * sizeof(float));
    hipMalloc((void **)&d_P, size * sizeof(float));

    hipMemcpy(d_N, h_N, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, kernelSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(O_TILE_SIZE + 4, 1, 1);
    dim3 numBlocks(ceil(size / O_TILE_SIZE), 1, 1);

    tiled_conv_1d<<<threadsPerBlock, numBlocks>>>(d_N, d_M, d_P, size, kernelSize);

    hipMemcpy(h_P, d_P, size * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < size; i++)
    // {
    //     printf("%f ", h_P[i]);
    // }
    // printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

void test_tiled_conv_2d (){
    int width = 255, height = 255;
    int kernelSize = 5;
    float h_N[width * height];
    float h_M[kernelSize * kernelSize];
    float h_P[width * height];

    for (int i = 0; i < width * height; i++)
    {
        h_N[i] = static_cast<float>(i);
    }
    for (int i = 0; i < kernelSize * kernelSize; i++)
    {
        h_M[i] = static_cast<float>(i+3);
    }

    float *d_N, *d_M, *d_P;

    hipMalloc((void **)&d_N, width * height * sizeof(float));
    hipMalloc((void **)&d_M, kernelSize * kernelSize * sizeof(float));
    hipMalloc((void **)&d_P, width * height * sizeof(float));

    hipMemcpy(d_N, h_N, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(O_TILE_SIZE_2D + kernelSize - 1, O_TILE_SIZE_2D + kernelSize - 1, 1);
    dim3 numBlocks(ceil(width / O_TILE_SIZE_2D), ceil(height / O_TILE_SIZE_2D), 1);

    tiled_conv_2d<<<threadsPerBlock, numBlocks>>>(d_N, d_M, d_P, width, height, kernelSize);

    hipMemcpy(h_P, d_P, width * height * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < width * height; i++)
    {
        printf("%f ", h_P[i]);
    }
    printf("\n");

    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
}

int main()
{
    auto start = std::chrono::high_resolution_clock::now();
    test_simple_conv_1d();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("plain_conv_1d: %f\n", diff.count());

    start = std::chrono::high_resolution_clock::now();
    test_simple_conv_2d();
    end = std::chrono::high_resolution_clock::now();
    diff = end - start;
    printf("plain_conv_2d: %f\n", diff.count());

    start = std::chrono::high_resolution_clock::now();
    test_tiled_conv_1d();
    end = std::chrono::high_resolution_clock::now();
    diff = end - start;
    printf("tiled_conv_1d: %f\n", diff.count());

    start = std::chrono::high_resolution_clock::now();
    test_tiled_conv_2d();
    end = std::chrono::high_resolution_clock::now();
    diff = end - start;
    printf("tiled_conv_2d: %f\n", diff.count());
}
